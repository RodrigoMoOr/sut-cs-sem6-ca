﻿#include "hip/hip_runtime.h"


#include <stdio.h>

// Monika Dudzinska
// Borys Pala
// Rodrigo Morales


const int cudaBlockSize = 2;
const int arraySize = 6;

const int arraySizeA = 17;

hipError_t addWithCuda(int* c, int* a, int* b, unsigned int size, unsigned int aSize);
hipError_t addWithCudaMatrices(int c[][arraySize], int a[][arraySize], int b[][arraySize]);

__global__ void addKernel(int* c, int* a, int* b)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void addKernelMatrices(int c[][arraySize], const int a[][arraySize], const int b[][arraySize], unsigned int arraySize)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i < arraySize) && (y < arraySize))
        c[i][y] = a[i][y] + b[i][y];
}

int main()
{
    int a[arraySizeA] = { 1,2,3,4,5,6, 7 ,8,9,10,11,12,13,14,15,16,17};
    int b[arraySizeA] = { 110,120,130,140,150,160,170, 180,190,200,210,220,230,240,250,260,270 };
    int c[arraySizeA] = { 0 };

    int a_two[arraySize][arraySize] = { 0 };
    int b_two[arraySize][arraySize] = { 0 };
    int c_two[arraySize][arraySize] = { 0 };

  

    for (size_t i = 0; i < arraySize; i++)
    {
        for (size_t y = 0; y < arraySize; y++)
        {
            a_two[i][y] = i + 1;
            b_two[i][y] = (i + 1) * 10;
            c_two[i][y] = 0;
        }
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySizeA, cudaBlockSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    printf("{");
    for (size_t i = 0; i < arraySizeA; i++)
    {
        printf("%d, ", a[i]);
    }
    printf("} + {");
    for (size_t i = 0; i < arraySizeA; i++)
    {
        printf("%d, ", b[i]);
    }
    printf("} = {");
    for (size_t i = 0; i < arraySizeA; i++)
    {
        printf("%d, ", c[i]);
    }
    printf("}\n");

    // Add vectors in parallel.
    hipError_t cudaStatus_two = addWithCudaMatrices(c_two, a_two, b_two);
    if (cudaStatus_two != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("\n");

    for (size_t i = 0; i < arraySize; i++)
    {
        for (size_t y = 0; y < arraySize; y++)
        {
            printf("%d , ", a_two[i][y]);
        }
        printf("\n");
    }

    printf("\n");

    for (size_t i = 0; i < arraySize; i++)
    {
        for (size_t y = 0; y < arraySize; y++)
        {
            printf("%d , ", b_two[i][y]);
        }
        printf("\n");
    }

    printf("\n");

    for (size_t i = 0; i < arraySize; i++)
    {
        for (size_t y = 0; y < arraySize; y++)
        {
            printf("%d , ", c_two[i][y]);
        }
        printf("\n");
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


    return 0;
}

// Function to add 2 1D matrices with CUDA
hipError_t addWithCuda(int* c, int* a, int* b, unsigned int size, unsigned int bSize)
{

    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    unsigned int bCount;
    hipError_t cudaStatus;

    if (size % bSize) {
        bCount = size / bSize + 1;
    }
    else {
        bCount = size / bSize;
    }
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel <<<bCount, size >>> (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

// Function to add 2D matrices with CUDA
hipError_t addWithCudaMatrices(int c[][arraySize], int a[][arraySize], int b[][arraySize])
{
    int(*dev_a)[arraySize] = 0;
    int(*dev_b)[arraySize] = 0;
    int(*dev_c)[arraySize] = 0;
    hipError_t cudaStatus;

    unsigned int bCount;
    if (arraySize % cudaBlockSize) {
        bCount = arraySize / cudaBlockSize + 1;
    }
    else {
        bCount = arraySize / cudaBlockSize;
    }

    dim3 blocks(cudaBlockSize, cudaBlockSize);
    dim3 threads(bCount, bCount);


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, arraySize * arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, arraySize * arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, arraySize * arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, arraySize * arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, arraySize * arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernelMatrices <<<blocks, threads>>> (dev_c, dev_a, dev_b, arraySize);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, arraySize * arraySize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}